

#include <hip/hip_runtime.h>
#include <stdio.h>

static void HandleError( hipError_t err,  const char *file,     int line ) {
    if (err != hipSuccess) {
        ::printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line );
        ::exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define N 100000

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",msg, hipGetErrorString(__err),  __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void add( int *a, int *b, int *c ) {
    int tid = blockIdx.x;    // this thread handles the data at its thread id
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

extern "C" void Cuda_Main_Test1(){
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );
   // cudaCheckErrors("cudamalloc fail");
    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),    hipMemcpyHostToDevice ) );
    //HANDLE_ERROR( cudaMemcpy( dev_b, b, N * sizeof(int),    cudaMemcpyHostToDevice ) );

    hipError_t status = hipMemcpy(dev_b, b, N * sizeof(int),    hipMemcpyHostToDevice);
  //  cudaCheckErrors("cuda memcpy fail");
    //sleep(1);
    add<<<N,1>>>(dev_a,dev_b,dev_c);
   // sleep(1);
    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),    hipMemcpyDeviceToHost ) );
//    if (status == cudaSuccess) {
//         printf( "0 CUDA Error: %s \n", cudaGetErrorString( status ) );
//    }
//    if (status != cudaSuccess) {
//         printf( "1 CUDA Error:  %s  \n", cudaGetErrorString( status ) );
//       // cout << "CUDA Error: " << cudaGetErrorString( cuerr ) << endl
//    }
//    cudaCheckErrors("cudamemcpy or cuda kernel fail");
   // sleep(1);
    // display the results
    for (int i=0; i<N; i++) {
      //  printf( "%d + %d = %d\n", a[i], b[i], c[i] );
        printf( "%d%c ", c[i],((i%8==7)?'\n':' '));
    }

    printf( "Start CPU add \n ");
    for (int i=0; i<N; i++) {
     c[i]=a[i]+b[i];
        //printf( "%d%c ", c[i],((i%8==7)?'\n':' '));
    }
     printf( "Stop CPU add \n ");
    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

}
