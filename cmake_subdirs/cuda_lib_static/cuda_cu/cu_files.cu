#include "hip/hip_runtime.h"
#include "cu_files.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>

CU_files::CU_files(){
  std::cout << __PRETTY_FUNCTION__ << "\n";
  int device_CU = 0;

  hipError_t error_id = hipGetDeviceCount(&device_CU);

  if(error_id == hipSuccess ){
      std::cout << "device_CU: " << device_CU << "\n";

      for (int i = 0; i < device_CU; ++i) {
          hipSetDevice(i);
          hipDeviceProp_t cdp;

          if( (error_id = hipGetDeviceProperties(&cdp,i)) != hipSuccess )
            break;
          std::cout << "hipGetDeviceProperties: " << cdp.name << "\n";

          int drv_get_ver{0};
          if( (error_id = hipDriverGetVersion(&drv_get_ver)) != hipSuccess )
            break;
          std::cout << "hipDriverGetVersion: " <<drv_get_ver << "\n";

          int run_time_ver{0};
          if( (error_id = hipRuntimeGetVersion(&run_time_ver)) != hipSuccess )
            break;
          std::cout << "cudaRunTimeGetVersion: " <<run_time_ver << "\n";

        }

    }else{
      std::cout << " NO CUDA DEVICES \n";
    }

}

CU_files::~CU_files(){
    std::cout << __PRETTY_FUNCTION__ << "\n";
}


__global__ void kernel(){

}

__global__ void add(int a, int b, int* c){
  *c = a + b;
}

// выделить память на
// скопировать данные из памяти CPU в память GPU
// осуществить запуск ядра <<<N,N>>>
// скопировать результаты вычислений в память CPU
// освободить память GPU

bool CU_files::init(){
  hipError_t error_id ;

  int c;
  int *dev_c;

  if( (error_id = hipMalloc((void **)&dev_c, sizeof (int))) != hipSuccess )
    return error_id;

  kernel<<<1,1>>>();

  add<<<1,1>>>(2,7,dev_c);

  if( (error_id = hipMemcpy( &c, dev_c, sizeof (int), hipMemcpyDeviceToHost) ) != hipSuccess )
    return error_id;

  std::cout << " 2 + 7 = " << c << "\n";

  hipFree(dev_c);

  return 0;

}


void CU_files::start(){}

void CU_files::stop(){}

