#include "hip/hip_runtime.h"
#include "cuda_shared_lib.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>

Cuda_shared_lib::Cuda_shared_lib(int sz) :
  sz_( sz!=0 ? sz : SIZE)
{
  std::cout << __PRETTY_FUNCTION__ << "\n";
  int device_CU = 0;

  hipError_t error_id = hipGetDeviceCount(&device_CU);

  if(error_id == hipSuccess ){
      std::cout << "device_CU: " << device_CU << "\n";

      for (int i = 0; i < device_CU; ++i) {
          hipSetDevice(i);
          hipDeviceProp_t cdp;

          if( (error_id = hipGetDeviceProperties(&cdp,i)) != hipSuccess )
            break;
          std::cout << "hipGetDeviceProperties: " << cdp.name << "\n";

          int drv_get_ver{0};
          if( (error_id = hipDriverGetVersion(&drv_get_ver)) != hipSuccess )
            break;
          std::cout << "hipDriverGetVersion: " <<drv_get_ver << "\n";

          int run_time_ver{0};
          if( (error_id = hipRuntimeGetVersion(&run_time_ver)) != hipSuccess )
            break;
          std::cout << "cudaRunTimeGetVersion: " <<run_time_ver << "\n";

        }




    }else{
      std::cout << " NO CUDA DEVICES \n";
    }

}

Cuda_shared_lib::~Cuda_shared_lib(){
    std::cout << __PRETTY_FUNCTION__ << "\n";
}


__global__ void kernel(){

}

__global__ void add(int a, int b, int* c){
  *c = a + b;
}

__global__ void vec_add(int *a, int *b, int* c){

 //http://ssd.sscc.ru/sites/default/files/content/attach/332/cuda-2-program-arch.pdf
 // преобразование 2D координат в линейный номер
  int idx = blockIdx.x * blockDim.x  + threadIdx.x;
  int idy = blockIdx.y * blockDim.y  + threadIdx.y;

   c[idx] = a[idx] + b[idx];

}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void dot( int *a, int *b, int *c ) {

  __shared__ int temp[THREADS_PER_BLOCK];

  int index = threadIdx.x + blockIdx.x*blockDim.x;

  temp[threadIdx.x] = a[index] * b[index];

  __syncthreads();

  if( 0 == threadIdx.x ) {

      int sum = 0;

      for( int i = 0; i < THREADS_PER_BLOCK; i++ )
        sum += temp[i];

      *c += sum;

      atomicAdd( c , sum );

    }
}

// выделить память на
// скопировать данные из памяти CPU в память GPU
// осуществить запуск ядра <<<N,N>>>
// скопировать результаты вычислений в память CPU
// освободить память GPU

bool Cuda_shared_lib::init(){
  hipError_t error_id ;

  if( (error_id = hipMalloc((void **)&v_a, sizeof (int))) != hipSuccess )
    return error_id;
  if( (error_id = hipMalloc((void **)&v_b, sizeof (int))) != hipSuccess )
    return error_id;
  if( (error_id = hipMalloc((void **)&v_c, sizeof (int))) != hipSuccess )
    return error_id;

}

void Cuda_shared_lib::apply(){

  /// зачем делать так если есть define
  dim3 dim_grid{32};
  dim3 dim_block{32};

 dot<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(v_a,v_b,v_c);

}

bool Cuda_shared_lib::de_init(){
  hipFree(v_a);
  hipFree(v_b);
  hipFree(v_c);
}

void Cuda_shared_lib::start(){}

void Cuda_shared_lib::stop(){}


//#define N_BLOCKS 1024;
//#define N_THREADS 1024;

//template<class T1, class T2>
//struct prefetch :
//    public thrust::unary_function<T1,T2>
//{
//  const T1* data;
//  prefetch(T1* _data) :
//    data(_data)
//  {};

//  // This method prefetchs the previous grid of data point into the L2.
//  __device__  T1 operator()(T2 i) {

//    if( (i-N_BLOCKS*N_THREADS) > 0) { //prefetch the previous grid
//        const T1 *pt = &data[i − (N_BLOCKS*N_THREADS)];
//        asm volatile ("prefetch.global.L2 [%0];"::"l"(pt) );
//      }
//    return data[i];

//  }
//};

//hipError_t error_id ;

//int c;
//int *dev_c;

//if( (error_id = hipMalloc((void **)&dev_c, sizeof (int))) != hipSuccess )
//  return error_id;

//kernel<<<1,1>>>();

//add<<<1,1>>>(2,7,dev_c);

//if( (error_id = hipMemcpy( &c, dev_c, sizeof (int), hipMemcpyDeviceToHost) ) != hipSuccess )
//  return error_id;

//std::cout << " 2 + 7 = " << c << "\n";

//hipFree(dev_c);

//return 0;
